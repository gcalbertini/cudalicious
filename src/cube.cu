// https://www.youtube.com/watch?v=iaRs_yJA_js&list=PLAwxTw4SYaPnFKojVQrmyOGFCqHTxfdv2&index=38

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__ void cube(float * d_out, float * d_in){
	int idx = threadIdx.x;
	float f = d_in[idx];
	d_out[idx] = f * f * f;
}

int main(int argc, char ** argv){
	constexpr int ARRAY_SIZE {96};
	constexpr int ARRAY_BYTES {ARRAY_SIZE*sizeof(float)}; 

	float h_in[ARRAY_SIZE];

	for (int i = 0; i < ARRAY_SIZE; ++i){
		h_in[i] = float(i);
	}
	float h_out[ARRAY_SIZE];

	float * d_in, * d_out;

	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	hipMemcpy(d_in, h_in, ARRAY_SIZE, hipMemcpyHostToDevice);

	cube<<<1, ARRAY_SIZE>>>(d_out, d_in);

	hipMemcpy(h_out, d_out, ARRAY_SIZE, hipMemcpyDeviceToHost);

	// print out resulting array
	for (int i = 0; i < ARRAY_SIZE; ++i){
		printf("%f", h_out[i]);
		printf(((i % 4) != 3) ? "\t" : "\n");
	}

	hipFree(d_in);
	hipFree(d_out);

	return 0;


}